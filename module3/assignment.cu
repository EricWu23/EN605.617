//Based on the work of Andrew Krepps

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>
    
#ifndef ARRAY_SIZE_X  
    #define ARRAY_SIZE_X 512 // column of the 2D array// this can be defined in Makefile through commandline overide (-D flag for compiler)
#endif
    
#define ARRAY_SIZE_Y 1  //row of the 2D array

#define WARP 32
#define ARRAY_SIZE_IN_BYTES ((ARRAY_SIZE_X) * (ARRAY_SIZE_Y) * (sizeof(int)))
    
    

__global__ void arrayAdd(int *array0,int *array1,int* arraysum) {

    // collapse the higher dimension layout or nested layout down to flat 2D
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	
    // collapse flat 2D down to 1D, whose index is global thread index
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;
    
    if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
        arraysum[global_idx]=array0[global_idx]+array1[global_idx];
    }
}
__global__ void arraySubtract(int *array0,int *array1,int* arraysub) {

    // collapse the higher dimension layout or nested layout down to flat 2D
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	
    // collapse flat 2D down to 1D, whose index is global thread index
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;
    
    if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
        arraysub[global_idx]=array0[global_idx]-array1[global_idx];
    }
}
    
__global__ void arrayMult(int *array0,int *array1,int* arraymult) {

    // collapse the higher dimension layout or nested layout down to flat 2D
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

    // collapse flat 2D down to 1D, whose index is global thread index
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;

    if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
    arraymult[global_idx]=array0[global_idx]*array1[global_idx];
    }
}
    
__global__ void arrayMod(int *array0,int *array1,int* arraymod) {

    // collapse the higher dimension layout or nested layout down to flat 2D
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

    // collapse flat 2D down to 1D, whose index is global thread index
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;

    if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
    arraymod[global_idx]=array0[global_idx]%array1[global_idx];
    }
}    
    
// function to print out a 2D array for debugging
 void print_array(int arr[ARRAY_SIZE_Y][ARRAY_SIZE_X] )
{   
      for(int i=0; i<ARRAY_SIZE_X; i++)
      {
            for(int j=0; j<ARRAY_SIZE_Y; j++)
            {
              if (i== ARRAY_SIZE_X-1)
                {
                  printf("%i\n", arr[j][i]);
                }
              else
                {
                  printf("%i ", arr[j][i]);
                }
    
            }
         
     }
}


/* Declare  arrays on the cpu */

int cpu_array0 [ARRAY_SIZE_Y][ARRAY_SIZE_X];
int cpu_array1 [ARRAY_SIZE_Y][ARRAY_SIZE_X];
int cpu_arrayadd [ARRAY_SIZE_Y][ARRAY_SIZE_X];
int cpu_arraysubtract [ARRAY_SIZE_Y][ARRAY_SIZE_X];
int cpu_arraymult [ARRAY_SIZE_Y][ARRAY_SIZE_X];
int cpu_arraymod [ARRAY_SIZE_Y][ARRAY_SIZE_X];

/* initialize the data in the array according to assignment requirement*/
void cpu_array_int(int numBlocks,int blockSize){
 int totalThreads=numBlocks*blockSize;
    if(totalThreads!=ARRAY_SIZE_X*ARRAY_SIZE_Y)
    {
       printf("Total number of Threads specified from command line does not match total number of data elements in the array. Initialization failed\n");
       printf("The total number of elements in array is :%i\n",ARRAY_SIZE_X*ARRAY_SIZE_Y);
       printf("Either give a commandline argument that match the array size or recompile by modifying the macro defintion ARRAY_SIZE_X");
       exit(1);
    }
    else
    {
         for(int i=0; i<ARRAY_SIZE_Y; i++)
         {
                for(int j=0; j<ARRAY_SIZE_X; j++)
                {
                     cpu_array0[i][j]=i*ARRAY_SIZE_X+j;// the first array contain value from 0 to (totalThreads-1)
                     cpu_array1[i][j]=rand() % 4;// generate value of second array element as a random number between 0 and 3
                }    
         
         }
         printf("The following two arrays are initialized! \n");
         printf("Array0:\n");
         print_array(cpu_array0);
         printf("--------------------------------------------\n");

         printf("Array1:\n");
         print_array(cpu_array1);
        printf("--------------------------------------------\n");
    
    }
     
}

void main_sub0(int numBlocks,int blockSize)
{
    /* data init*/
    cpu_array_int(numBlocks,blockSize);
     
    /* layout specification
     1. assume that blockSize is at least 64 and will be multiple of 32
     2. numberBlocks will be at least 1
    */
    const dim3 threads_layout(WARP,blockSize/WARP); // there are multiple ways of layout to achieve blocksize. I choose to fix the  blockDim.x as the WARP size
    const dim3 blocks_layout(1,numBlocks);// there are multiple ways of layout to achieve numBlocks, I choose to fix the gridDim.x to 1
    
    /* Declare statically arrays */
    int * gpu_array0, * gpu_array1,*gpu_arrayresult;
    
    int size_in_bytes = ARRAY_SIZE_X* ARRAY_SIZE_Y* sizeof(int);
    
    // memory allocation on GPU
    hipMalloc((void **)&gpu_array0, size_in_bytes);
	hipMalloc((void **)&gpu_array1, size_in_bytes);
    hipMalloc((void **)&gpu_arrayresult, size_in_bytes);
    
    // memory copy from cpu to gpu
    hipMemcpy( gpu_array0,cpu_array0 , size_in_bytes, hipMemcpyHostToDevice );
    hipMemcpy( gpu_array1,cpu_array1 , size_in_bytes, hipMemcpyHostToDevice );
  
    for(int kernel=0; kernel<4; kernel++)
    {
      switch(kernel)
      {
            case 0:{
                    auto start = std::chrono::high_resolution_clock::now();   
                    arrayAdd<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult); // kernel call to add two 2-D arrays
    
                    auto stop = std::chrono::high_resolution_clock::now();
                                
                    hipMemcpy(cpu_arrayadd, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
                    
                     printf("Kernel 0 (Add) is called! \n");
                     printf("Array Result:\n");
                     print_array(cpu_arrayadd);
                     std::cout << "Time taken by GPU: " << std::chrono::duration_cast<std::chrono::nanoseconds>(stop-start).count() << "ns\n";
                     printf("--------------------------------------------\n");
                    } break;
                        
                                
                                
            case 1:{
                    auto start = std::chrono::high_resolution_clock::now();   
                    arraySubtract<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult);//kernel call to subtract two 2-D arrays 
                    auto stop = std::chrono::high_resolution_clock::now();
    
                     hipMemcpy(cpu_arraysubtract, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
                     
                        printf("Kernel 1 (subtract) is called! \n");
                         printf("Array Result:\n");
                         print_array(cpu_arraysubtract);
                        std::cout << "Time taken by GPU: " << std::chrono::duration_cast<std::chrono::nanoseconds>(stop-start).count() << "ns\n";  
                         printf("--------------------------------------------\n");
                   }break;                    
                                
                                
           case 2:{
    
                    auto start = std::chrono::high_resolution_clock::now(); 
                     arrayMult<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult);//kernel call to (elementwise)multiply two 2-D arrays 
                    auto stop = std::chrono::high_resolution_clock::now();
    
                     hipMemcpy(cpu_arraymult, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
    
    
                     printf("Kernel 2 (multiplication) is called! \n");
                     printf("Array Result:\n");
                     print_array(cpu_arraymult);  
                    std::cout << "Time taken by GPU: " << std::chrono::duration_cast<std::chrono::nanoseconds>(stop-start).count() << "ns\n";
                     printf("--------------------------------------------\n");
                   }break;                    
                                               
           case 3:{      
                    auto start = std::chrono::high_resolution_clock::now(); 
                     arrayMod<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult);//kernel call to (elementwise) mod divide two 2-D arrays 
                    auto stop = std::chrono::high_resolution_clock::now();
                     hipMemcpy(cpu_arraymod, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
                     printf("Kernel 3 (mod) is called! \n");
                     printf("Array Result:\n");
                     print_array(cpu_arraymod);   
                     std::cout << "Time taken by GPU: " << std::chrono::duration_cast<std::chrono::nanoseconds>(stop-start).count() << "ns\n";
                     printf("--------------------------------------------\n");
                   }break;                    
                                               
            default: exit(1); break;                    
                                                             
                                
      }
      
     
                                
                              
    }
                                
    /* Free the arrays on the GPU as now we're done with them */
    hipFree(gpu_array0);
	hipFree(gpu_array1);
    hipFree(gpu_arrayresult);

}




int main(int argc, char** argv)
{
	// read command line arguments
	int totalThreads = (1 << 20);
	int blockSize = 256;
	
	if (argc >= 2) {
		totalThreads = atoi(argv[1]);
        printf("Total number of threads changed to:%i\n", totalThreads);
	}
	if (argc >= 3) {
		blockSize = atoi(argv[2]);
        printf("number of thread per block changed to:%i\n", blockSize);
	}

	int numBlocks = totalThreads/blockSize;
    
    
    /* code check to make sure blockSize is multiple of WARP */
    if(blockSize<WARP){                        
           blockSize=WARP;
           printf("Warning: Block size specified is less than size of WARP.It got modified to be: %i\n",WARP);     
         }
     else{
            if(blockSize % WARP!=0)
            {
                    blockSize=(blockSize+0.5*WARP)/WARP*WARP;
                    printf("Warning: Block size specified is not evenly divisible by the size of WARP.\n");
                    //printf("It got modified to be the nearst number that can be evenly divisible by the size of WARP.\n");
                    // printf("Now, the blocksize is:%i\n",blockSize);     
            }
         }
    
	// validate command line arguments
	if (totalThreads % blockSize != 0) {
		++numBlocks;
		totalThreads = numBlocks*blockSize;
		
		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}
    
    main_sub0(numBlocks,blockSize);

}
