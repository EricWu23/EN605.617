/* 
	File Description: 
	This file explores the impact of warp divergence based on simple operations (add, subtract)
	The plan is 
	1. we have two 1x32 array on the host
	2. copy the array from the host memory into GPU memory 
	3. Create three different kernels,
		3.1 kernel1: if the global index is even, perform add, if the index is odd, perform subtract
		3.2 kernel2: if the global index is larger than 15, subtract. If the global index is less or equal to 15, add
		3.3 kernel3: if the global index is less than 10, add. if the global index is between 10 and 25, subtract. if the global index is larger than 25,add  
    4. Time the operation of all the three kernels and see if there is any difference.
	
	The intension is that in all three cases, we are executing 16 adds and 16 subtracts of two ints. However, due to the different way we branch the operation, we expect to 
	obtain different execution speed.
	
	Author: Yujiang Wu
	Date: 2022/02/14

*/ 

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>

#ifndef ARRAY_SIZE_X  
    #define ARRAY_SIZE_X 32 // column of the 2D array// this can be defined in Makefile through commandline overide (-D flag for compiler)
#endif
    
#define ARRAY_SIZE_Y 1  //row of the 2D array

#define WARP 32
#define ARRAY_SIZE_IN_BYTES ((ARRAY_SIZE_X) * (ARRAY_SIZE_Y) * (sizeof(int)))
    
__global__ void Kernel1(int *array0,int *array1,int* arrayresult) {
	  
       // collapse the higher dimension layout or nested layout down to flat 2D
		const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
		const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	   // collapse flat 2D down to 1D, whose index is global thread index
		const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;
	if(global_idx%2==0)
	{
		arrayresult[global_idx]=array0[global_idx]+array1[global_idx];
	}	
	else{
		arrayresult[global_idx]=array0[global_idx]-array1[global_idx];	
	}                        
}    
__global__ void Kernel2(int *array0,int *array1,int* arrayresult) {
	  
       // collapse the higher dimension layout or nested layout down to flat 2D
		const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
		const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	   // collapse flat 2D down to 1D, whose index is global thread index
		const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;
	if(global_idx>15)
	{
		arrayresult[global_idx]=array0[global_idx]+array1[global_idx];
	}	
	else
	{
		arrayresult[global_idx]=array0[global_idx]-array1[global_idx];	
	}                        
}    

__global__ void Kernel3(int *array0,int *array1,int* arrayresult) {
	  
       // collapse the higher dimension layout or nested layout down to flat 2D
		const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
		const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	   // collapse flat 2D down to 1D, whose index is global thread index
		const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;
	if(global_idx<10)
	{
		arrayresult[global_idx]=array0[global_idx]+array1[global_idx];
	}	
	else if(global_idx>=10 && global_idx<=25)
	{
		arrayresult[global_idx]=array0[global_idx]-array1[global_idx];	
	}
	else{
		arrayresult[global_idx]=array0[global_idx]+array1[global_idx];
	}
}    
    
// function to print out a 2D array for debugging
 void print_array(int arr[ARRAY_SIZE_Y][ARRAY_SIZE_X] )
{   
      for(int i=0; i<ARRAY_SIZE_X; i++)
      {
            for(int j=0; j<ARRAY_SIZE_Y; j++)
            {
              if (i== ARRAY_SIZE_X-1)
                {
                  printf("%i\n", arr[j][i]);
                }
              else
                {
                  printf("%i ", arr[j][i]);
                }
    
            }
         
     }
}


/* Declare  arrays on the cpu */

int cpu_array0 [ARRAY_SIZE_Y][ARRAY_SIZE_X];
int cpu_array1 [ARRAY_SIZE_Y][ARRAY_SIZE_X];
int cpu_result [ARRAY_SIZE_Y][ARRAY_SIZE_X];

/* initialize the data in the array according to assignment requirement*/
void cpu_array_int(int numBlocks,int blockSize){
 int totalThreads=numBlocks*blockSize;
    if(totalThreads!=ARRAY_SIZE_X*ARRAY_SIZE_Y)
    {
       printf("Total number of Threads specified from command line does not match total number of data elements in the array. Initialization failed\n");
       printf("The total number of elements in array is :%i\n",ARRAY_SIZE_X*ARRAY_SIZE_Y);
       printf("Either give a commandline argument that match the array size or recompile by modifying the macro defintion ARRAY_SIZE_X");
       exit(1);
    }
    else
    {
         for(int i=0; i<ARRAY_SIZE_Y; i++)
         {
                for(int j=0; j<ARRAY_SIZE_X; j++)
                {
                     cpu_array0[i][j]=i*ARRAY_SIZE_X+j;// the first array contain value from 0 to (totalThreads-1)
                     cpu_array1[i][j]=rand() % 4;// generate value of second array element as a random number between 0 and 3
                }    
         
         }
         printf("The following two arrays are initialized! \n");
         printf("Array0:\n");
         print_array(cpu_array0);
         printf("--------------------------------------------\n");

         printf("Array1:\n");
         print_array(cpu_array1);
        printf("--------------------------------------------\n");
    
    }
     
}

void main_sub0(int numBlocks,int blockSize)
{
    /* data init*/
    cpu_array_int(numBlocks,blockSize);
     
    /* layout specification
     1. assume that blockSize is at least 64 and will be multiple of 32
     2. numberBlocks will be at least 1
    */
    const dim3 threads_layout(WARP,blockSize/WARP); // there are multiple ways of layout to achieve blocksize. I choose to fix the  blockDim.x as the WARP size
    const dim3 blocks_layout(1,numBlocks);// there are multiple ways of layout to achieve numBlocks, I choose to fix the gridDim.x to 1
    
    /* Declare statically arrays */
    int * gpu_array0, * gpu_array1,*gpu_arrayresult;
    
    int size_in_bytes = ARRAY_SIZE_X* ARRAY_SIZE_Y* sizeof(int);
    
    // memory allocation on GPU
    hipMalloc((void **)&gpu_array0, size_in_bytes);
	hipMalloc((void **)&gpu_array1, size_in_bytes);
    hipMalloc((void **)&gpu_arrayresult, size_in_bytes);
    
    // memory copy from cpu to gpu
    hipMemcpy( gpu_array0,cpu_array0 , size_in_bytes, hipMemcpyHostToDevice );
    hipMemcpy( gpu_array1,cpu_array1 , size_in_bytes, hipMemcpyHostToDevice );
                                             
  auto start = std::chrono::high_resolution_clock::now();
  for(int k=0;k<1000;k++)
  { 
	Kernel1<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult);
  }   
  auto stop = std::chrono::high_resolution_clock::now();
	
  hipMemcpy(cpu_result, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
 printf("Kernel1 is called 1000 times! \n");
			printf("Array Result:\n");
			print_array(cpu_result);
			std::cout << "Total Time taken by GPU: " << std::chrono::duration_cast<std::chrono::nanoseconds>(stop-start).count() << "ns\n";
			printf("--------------------------------------------\n");
	 
    start = std::chrono::high_resolution_clock::now();
    for(int k=0;k<1000;k++)
    {
        Kernel2<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult);
	}
    stop = std::chrono::high_resolution_clock::now();
	
    hipMemcpy(cpu_result, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
	printf("Kernel2 is called 1000 times! \n");
			printf("Array Result:\n");
			print_array(cpu_result);
			std::cout << "Total Time taken by GPU: " << std::chrono::duration_cast<std::chrono::nanoseconds>(stop-start).count() << "ns\n";
			printf("--------------------------------------------\n");

    start = std::chrono::high_resolution_clock::now();
    for(int k=0;k<1000;k++)
    {
        Kernel3<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult);
	}
    stop = std::chrono::high_resolution_clock::now();
	
    hipMemcpy(cpu_result, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
	printf("Kernel3 is called 1000 times! \n");
			printf("Array Result:\n");
			print_array(cpu_result);
			std::cout << "Total Time taken by GPU: " << std::chrono::duration_cast<std::chrono::nanoseconds>(stop-start).count() << "ns\n";
			printf("--------------------------------------------\n");

   
    
    /* Free the arrays on the GPU as now we're done with them */
    hipFree(gpu_array0);
	hipFree(gpu_array1);
    hipFree(gpu_arrayresult);

}




int main(int argc, char** argv)
{
	// read command line arguments
	int totalThreads = (1 << 20);
	int blockSize = 256;
	
	if (argc >= 2) {
		totalThreads = atoi(argv[1]);
        printf("Total number of threads changed to:%i\n", totalThreads);
	}
	if (argc >= 3) {
		blockSize = atoi(argv[2]);
        printf("number of thread per block changed to:%i\n", blockSize);
	}

	int numBlocks = totalThreads/blockSize;
    
    
    /* code check to make sure blockSize is multiple of WARP */
    if(blockSize<WARP){                        
           blockSize=WARP;
           printf("Warning: Block size specified is less than size of WARP.It got modified to be: %i\n",WARP);     
         }
     else{
            if(blockSize % WARP!=0)
            {
                    blockSize=(blockSize+0.5*WARP)/WARP*WARP;
                    printf("Warning: Block size specified is not evenly divisible by the size of WARP.\n");
                    //printf("It got modified to be the nearst number that can be evenly divisible by the size of WARP.\n");
                    // printf("Now, the blocksize is:%i\n",blockSize);     
            }
         }
    
	// validate command line arguments
	if (totalThreads % blockSize != 0) {
		++numBlocks;
		totalThreads = numBlocks*blockSize;
		
		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}
    
    main_sub0(numBlocks,blockSize);

}
