#include <hipsolver.h>
#include <stdio.h>
#include <assert.h>
#include <hip/hip_runtime.h>
#include "linearsolve.h"

void GPU_LinearSolve(float const *A,const int ht_A,const int wd_A, float const *B,const int nrhs,float const *X,hipStream_t stream){
	
	hipsolverHandle_t cusolverH = NULL;
	hipsolverStatus_t cusolver_status = HIPSOLVER_STATUS_SUCCESS;
    hipError_t cudaStat1 = hipSuccess;
    hipError_t cudaStat2 = hipSuccess;
    hipError_t cudaStat3 = hipSuccess;
    hipError_t cudaStat4 = hipSuccess;
    const int m = ht_A;
	const int n = wd_A;
	int info_gpu = 0;
    int *devInfo = NULL; // info in gpu (device copy)	
	float *d_work = NULL;
	size_t  lwork_bytes = 0;
	int niter=0;
// step 1: create cusolver/cublas handle
	 cusolver_status = hipsolverDnCreate(&cusolverH);
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
// step 2: copy A and B to device
	float *d_A,*d_B,*d_X= NULL;
	cudaStat1 = hipMalloc ((void**)&d_A  ,sizeof(float) * m * n);
	cudaStat2 = hipMalloc ((void**)&d_B  ,sizeof(float) * m * nrhs);
	cudaStat3 = hipMalloc ((void**)&d_X  , sizeof(float) * n * nrhs);
	cudaStat4 = hipMalloc ((void**)&devInfo,sizeof(int));

	assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
    assert(hipSuccess == cudaStat3);
    assert(hipSuccess == cudaStat4);
	cudaStat1 = hipMemcpy(d_A, A, sizeof(float) * m * n , hipMemcpyHostToDevice);
    cudaStat2 = hipMemcpy(d_B, B, sizeof(float) * m * nrhs, hipMemcpyHostToDevice);
	assert(hipSuccess == cudaStat1);
    assert(hipSuccess == cudaStat2);
// step 3: query and allocate working space for gels
	hipsolverSetStream(cusolverH,stream);
	cusolver_status=hipsolverDnSSgels_bufferSize(
						cusolverH,
						m,
						n,
						nrhs,
						d_A,
						m,
						d_B,
						m,
						d_X,
						n,
						d_work,
						&lwork_bytes);
	assert (cusolver_status == HIPSOLVER_STATUS_SUCCESS);
    
	cudaStat1 = hipMalloc((void**)&d_work,lwork_bytes);
    assert(hipSuccess == cudaStat1);
// step 4: linear solve
	hipsolverSetStream(cusolverH,stream);// set the kernel to run on the stream specified by stream
	cusolver_status=hipsolverDnSSgels(
						cusolverH,
						m,
						n,
						nrhs,
						d_A,
						m,
						d_B,
						m,
						d_X,
						n,
						d_work,
						lwork_bytes,
						&niter,
						devInfo);
	cudaStat1 = hipDeviceSynchronize();
    assert(HIPSOLVER_STATUS_SUCCESS == cusolver_status);
    assert(hipSuccess == cudaStat1);
    // check if linear solve is good or not
    cudaStat1 = hipMemcpy(&info_gpu, devInfo, sizeof(int), hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);
    printf("\nafter gels: info_gpu = %d\n", info_gpu);
    assert(0 == info_gpu);
	printf("\nafter gels: niter = %d\n", niter);
// step 5: Copy result to Host
    cudaStat1 = hipMemcpy((void*)X, d_X, sizeof(float)*n*nrhs, hipMemcpyDeviceToHost);
    assert(hipSuccess == cudaStat1);

// step 6: clean up
	if (d_A    ) hipFree(d_A);
    if (d_B    ) hipFree(d_B);
    if (devInfo) hipFree(devInfo);
    if (d_work ) hipFree(d_work);
	if (d_X    ) hipFree(d_X);

    if (cusolverH) hipsolverDnDestroy(cusolverH);   

    //hipDeviceReset();// comment out just to time the code
}
