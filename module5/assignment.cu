#include "hip/hip_runtime.h"
//Based on the work of Andrew Krepps
#include <stdio.h>
#include <stdlib.h>
#include <chrono>
#include <iostream>
#include "assignment.h"
    
#define WARP 32
#define OFFSET 10
#ifndef VERBOSE 
	#define VERBOSE 1
#endif

__global__ void arrayAdd(int *array0,int *array1,int* arraysum) {

    // collapse the higher dimension layout or nested layout down to flat 2D
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	
    // collapse flat 2D down to 1D, whose index is global thread index
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;
    
    if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
        arraysum[global_idx]=array0[global_idx]+array1[global_idx];
    }
}
__global__ void gpu_arrayAdd_shared(int *array0,int *array1,int* arraysum,int num_elements) {

    // collapse the higher dimension layout or nested layout down to flat 2D
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	
    // collapse flat 2D down to 1D, whose index is global thread index
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;
    
	 extern __shared__ int shared_tmp[];// total dynamically  allocated shared mem
	 int *arry0shared=shared_tmp;
	 int *arry1shared=(int*)&shared_tmp[num_elements];
	 
	 copy_data_to_shared(array0,arry0shared,num_elements,global_idx);
	 copy_data_to_shared(array1,arry1shared,num_elements,global_idx);
	 
    if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
        arraysum[global_idx]=arry0shared[global_idx]+arry1shared[global_idx];
    }
}
__global__ void arraySubtract(int *array0,int *array1,int* arraysub) {

    // collapse the higher dimension layout or nested layout down to flat 2D
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
	const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	
    // collapse flat 2D down to 1D, whose index is global thread index
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;
    
    if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
        arraysub[global_idx]=array0[global_idx]-array1[global_idx];
    }
}

__global__ void arrayMult(int *array0,int *array1,int* arraymult) {

    // collapse the higher dimension layout or nested layout down to flat 2D
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

    // collapse flat 2D down to 1D, whose index is global thread index
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;

    if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
    arraymult[global_idx]=array0[global_idx]*array1[global_idx];
    }
}

__global__ void arrayMod(int *array0,int *array1,int* arraymod) {

    // collapse the higher dimension layout or nested layout down to flat 2D
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;
    const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;

    // collapse flat 2D down to 1D, whose index is global thread index
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;

    if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
    arraymod[global_idx]=array0[global_idx]%array1[global_idx];
    }
}    

__host__ void execute_gpu_arrayAdd(int numBlocks,int blockSize,int *const gpu_array0,int * const gpu_array1,int* const gpu_arrayresult,int* const cpu_array_res)
{
	int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	
	/* layout specification*/
	const dim3 threads_layout(WARP,blockSize/WARP); // there are multiple ways of layout to achieve blocksize. I choose to fix the  blockDim.x as the WARP size
    const dim3 blocks_layout(1,numBlocks);// there are multiple ways of layout to achieve numBlocks, I choose to fix the gridDim.x to 1
	
	hipEvent_t kernel_start1, kernel_stop1;
	float delta_time1 = 0.0f;
	hipEventCreate(&kernel_start1);
	hipEventCreateWithFlags(&kernel_stop1,hipEventBlockingSync);
	
	
	//record events around kernel launch
	hipEventRecord(kernel_start1, 0);//0 is the default stream
	arrayAdd<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult); // kernel call to add two 2-D arrays 
	hipEventRecord(kernel_stop1, 0);
	hipEventSynchronize(kernel_stop1);
	hipEventElapsedTime(&delta_time1, kernel_start1,kernel_stop1);	
	
	hipMemcpy(cpu_array_res, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
	printf("Kernel 0 (Add) is called! \n");
	if(VERBOSE)
	{
		printf("Array Result:\n");
		print_array(cpu_array_res,cpu_arr_size_y,cpu_arr_size_x);					 
	}
	printf("GPU execution with global mem takes: %.3fms",delta_time1);
	printf("--------------------------------------------\n");
	
	hipEventDestroy(kernel_start1);
	hipEventDestroy(kernel_stop1);
}

__host__ void execute_gpu_arraySubtract(int numBlocks,int blockSize,int *const gpu_array0,int * const gpu_array1,int* const gpu_arrayresult,int* const cpu_array_res)
{
	int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	/* layout specification*/
	const dim3 threads_layout(WARP,blockSize/WARP); // there are multiple ways of layout to achieve blocksize. I choose to fix the  blockDim.x as the WARP size
    const dim3 blocks_layout(1,numBlocks);// there are multiple ways of layout to achieve numBlocks, I choose to fix the gridDim.x to 1
	hipEvent_t kernel_start1, kernel_stop1;
	float delta_time1 = 0.0f;
	hipEventCreate(&kernel_start1);
	hipEventCreateWithFlags(&kernel_stop1,hipEventBlockingSync);
	
	hipEventRecord(kernel_start1, 0);//0 is the default stream
	arraySubtract<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult);//kernel call to subtract two 2-D arrays 
	hipEventRecord(kernel_stop1, 0);
	hipEventSynchronize(kernel_stop1);
	hipEventElapsedTime(&delta_time1, kernel_start1,kernel_stop1);
	hipMemcpy(cpu_array_res, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
	printf("Kernel 1 (subtract) is called! \n");
	if(VERBOSE){
		printf("Array Result:\n");
		print_array(cpu_array_res,cpu_arr_size_y,cpu_arr_size_x);					 
	 }
	printf("GPU execution with global mem takes: %.3fms",delta_time1);
	printf("--------------------------------------------\n");
	
	hipEventDestroy(kernel_start1);
	hipEventDestroy(kernel_stop1);
}


__host__ void execute_gpu_arrayMult(int numBlocks,int blockSize,int *const gpu_array0,int * const gpu_array1,int* const gpu_arrayresult,int* const cpu_array_res)
{
    int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	/* layout specification*/
	const dim3 threads_layout(WARP,blockSize/WARP); // there are multiple ways of layout to achieve blocksize. I choose to fix the  blockDim.x as the WARP size
    const dim3 blocks_layout(1,numBlocks);// there are multiple ways of layout to achieve numBlocks, I choose to fix the gridDim.x to 1
	hipEvent_t kernel_start1, kernel_stop1;
	float delta_time1 = 0.0f;
	hipEventCreate(&kernel_start1);
	hipEventCreateWithFlags(&kernel_stop1,hipEventBlockingSync);
	
	hipEventRecord(kernel_start1, 0);//0 is the default stream
	arrayMult<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult);//kernel call to (elementwise)multiply two 2-D arrays 
	hipEventRecord(kernel_stop1, 0);//0 is the default stream
	hipEventSynchronize(kernel_stop1);
	hipEventElapsedTime(&delta_time1, kernel_start1,kernel_stop1);
	hipMemcpy(cpu_array_res, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
	printf("Kernel 2 (multiplication) is called! \n");
	if(VERBOSE){
		 printf("Array Result:\n");
		 print_array(cpu_array_res,cpu_arr_size_y,cpu_arr_size_x); 
	}
	printf("GPU execution with global mem takes: %.3fms",delta_time1);
	printf("--------------------------------------------\n");
	
	hipEventDestroy(kernel_start1);
	hipEventDestroy(kernel_stop1);	
}

__host__ void execute_gpu_arrayMod(int numBlocks,int blockSize,int *const gpu_array0,int * const gpu_array1,int* const gpu_arrayresult,int* const cpu_array_res)
{
    int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	/* layout specification*/
	const dim3 threads_layout(WARP,blockSize/WARP); // there are multiple ways of layout to achieve blocksize. I choose to fix the  blockDim.x as the WARP size
    const dim3 blocks_layout(1,numBlocks);// there are multiple ways of layout to achieve numBlocks, I choose to fix the gridDim.x to 1
	hipEvent_t kernel_start1, kernel_stop1;
	float delta_time1 = 0.0f;
	hipEventCreate(&kernel_start1);
	hipEventCreateWithFlags(&kernel_stop1,hipEventBlockingSync);
	
	hipEventRecord(kernel_start1, 0);//0 is the default stream
	arrayMod<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult);//kernel call to (elementwise) mod divide two 2-D arrays
	hipEventRecord(kernel_stop1, 0);//0 is the default stream
	hipEventSynchronize(kernel_stop1);
	hipEventElapsedTime(&delta_time1, kernel_start1,kernel_stop1);						
	hipMemcpy(cpu_array_res, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu 
	printf("Kernel 3 (mod) is called! \n");
	if(VERBOSE){
		printf("Array Result:\n");
		print_array(cpu_array_res,cpu_arr_size_y,cpu_arr_size_x);  
	}
	printf("GPU execution with global mem takes: %.3fms",delta_time1);
	printf("--------------------------------------------\n");
	
	hipEventDestroy(kernel_start1);
	hipEventDestroy(kernel_stop1);	
}

// function to print out a 2D array for debugging
void print_array(int* arr, int num_row, int num_col)
{
      printf("--------------------------------------------\n");
      for(int i=0; i<num_col; i++){
            for(int j=0; j<num_row; j++){
              if (i== num_col-1){
                  printf("%i\n", arr[j*num_col+i]);
              }
              else{
				  printf("%i ", arr[j*num_col+i]);
              }
            }
      }
      printf("--------------------------------------------\n");      
}


/* initialize the data in the array according to assignment requirement*/
void cpu_array0_int(int* arr,int num_row,int num_column){		
	 //2D array intialization 		
	 for(int i=0; i<num_row; i++){
			for(int j=0; j<num_column; j++){		
				 arr[i*num_column+j]=i*num_column+j;
			}    
	 }				
}


/* initialize the data in the array according to assignment requirement*/
void  cpu_array1_int(int* arr,int num_row,int num_column){	
	 //2D array intialization 		
	 for(int i=0; i<num_row; i++){
			for(int j=0; j<num_column; j++){
				 //arr[i][j]=i*num_column+j;// the first array contain value from 0 to (totalThreads-1)
				 arr[i*num_column+j]=rand() % 4;// generate value of second array element as a random number between 0 and 3
			}    	 
	 }				
}





__device__ void copy_data_to_shared(const int * const data,
									int * const shared_tmp,
									const int num_elements,
									const int tid)
{
	// deepcopy
	if(tid<num_elements)
	{
		shared_tmp[tid] = data[tid];
	}
	__syncthreads();// synchronize all the threads within a block
}

__host__ void execute_gpu_sharedmem_arrayAdd(int numBlocks,int blockSize,int *const gpu_array0,int * const gpu_array1,int* const gpu_arrayresult,int* const cpu_array_res)
{
	int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	
	/* layout specification*/
	const dim3 threads_layout(WARP,blockSize/WARP); // there are multiple ways of layout to achieve blocksize. I choose to fix the  blockDim.x as the WARP size
    const dim3 blocks_layout(1,numBlocks);// there are multiple ways of layout to achieve numBlocks, I choose to fix the gridDim.x to 1
	
	hipEvent_t kernel_start1, kernel_stop1;
	float delta_time1 = 0.0f;
	hipEventCreate(&kernel_start1);
	hipEventCreateWithFlags(&kernel_stop1,hipEventBlockingSync);
	
	
	//record events around kernel launch
	hipEventRecord(kernel_start1, 0);//0 is the default stream
	//arrayAdd<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult); // kernel call to add two 2-D arrays 
	gpu_arrayAdd_shared<<<blocks_layout,threads_layout,totalThreads*2*sizeof(int)>>>(gpu_array0,gpu_array1,gpu_arrayresult,totalThreads);
	hipEventRecord(kernel_stop1, 0);
	hipEventSynchronize(kernel_stop1);
	hipEventElapsedTime(&delta_time1, kernel_start1,kernel_stop1);	
	
	hipMemcpy(cpu_array_res, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
	printf("Kernel 0 (Add) is called! \n");
	if(VERBOSE)
	{
		printf("Array Result:\n");
		print_array(cpu_array_res,cpu_arr_size_y,cpu_arr_size_x);					 
	}
	printf("GPU execution with global mem takes: %.3fms",delta_time1);
	printf("--------------------------------------------\n");
	
	hipEventDestroy(kernel_start1);
	hipEventDestroy(kernel_stop1);
}

void execute_gpu_global_test(int numBlocks, int blockSize){
	int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);

	/* dynamically allocate the memory on the host*/
	int *cpu_array0,*cpu_array1,*cpu_array_res; 
	cpu_array0 = (int *) malloc(size_in_bytes);//pagable
	cpu_array1 = (int *)malloc(size_in_bytes);
	cpu_array_res = (int *)malloc(size_in_bytes);
	/* data init*/
    cpu_array0_int(cpu_array0,cpu_arr_size_y,cpu_arr_size_x);
	cpu_array1_int(cpu_array1,cpu_arr_size_y,cpu_arr_size_x);
	/* print out the arrays for debuging */
	if(VERBOSE){
		printf("The following two arrays are initialized on cpu! \n");
		printf("Array0:\n");
		print_array(cpu_array0,cpu_arr_size_y,cpu_arr_size_x);
		printf("Array1:\n");
		print_array(cpu_array1,cpu_arr_size_y,cpu_arr_size_x);
	}
	
	 /* Device memory allocation */
    int * gpu_array0, * gpu_array1,*gpu_arrayresult;
	hipMalloc((void **)&gpu_array0, size_in_bytes);
	hipMalloc((void **)&gpu_array1, size_in_bytes);
    hipMalloc((void **)&gpu_arrayresult, size_in_bytes);
	/* explicit memory copy from cpu to device*/
	hipMemcpy( gpu_array0,cpu_array0 , size_in_bytes, hipMemcpyHostToDevice );
    hipMemcpy( gpu_array1,cpu_array1 , size_in_bytes, hipMemcpyHostToDevice );
	
	/* Execute 4 simple math operation*/ 
	for(int kernel=0; kernel<4; kernel++)
    {
      switch(kernel)
      {
            case 0:{ 
					  execute_gpu_arrayAdd(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                    } break;                                                                                     
            case 1:{
					  execute_gpu_arraySubtract(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                   }break;                                     
           case 2:{
					  execute_gpu_arrayMult(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                   }break;                                                                 
           case 3:{      
					  execute_gpu_arrayMod(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                   }break;                                                                   
            default: exit(1); break;                                                                                                         
      }	
	}	
		
	/*Free the arrays on the CPU*/
	free(cpu_array0);
	free(cpu_array1);
	free(cpu_array_res);
    /* Free the arrays on the GPU as now we're done with them */
    hipFree(gpu_array0);
	hipFree(gpu_array1);
    hipFree(gpu_arrayresult);	
	//Destroy all allocations and reset all state on the current device in the current process
	hipDeviceReset();
}
void execute_gpu_shared_test(int numBlocks, int blockSize){
	int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);

	/* dynamically allocate the memory on the host*/
	int *cpu_array0,*cpu_array1,*cpu_array_res; 
	cpu_array0 = (int *) malloc(size_in_bytes);//pagable
	cpu_array1 = (int *)malloc(size_in_bytes);
	cpu_array_res = (int *)malloc(size_in_bytes);
	/* data init*/
    cpu_array0_int(cpu_array0,cpu_arr_size_y,cpu_arr_size_x);
	cpu_array1_int(cpu_array1,cpu_arr_size_y,cpu_arr_size_x);
	/* print out the arrays for debuging */
	if(VERBOSE){
		printf("The following two arrays are initialized on cpu! \n");
		printf("Array0:\n");
		print_array(cpu_array0,cpu_arr_size_y,cpu_arr_size_x);
		printf("Array1:\n");
		print_array(cpu_array1,cpu_arr_size_y,cpu_arr_size_x);
	}
	
	 /* Device memory allocation */
    int * gpu_array0, * gpu_array1,*gpu_arrayresult;
	hipMalloc((void **)&gpu_array0, size_in_bytes);
	hipMalloc((void **)&gpu_array1, size_in_bytes);
    hipMalloc((void **)&gpu_arrayresult, size_in_bytes);
	/* explicit memory copy from cpu to device*/
	hipMemcpy( gpu_array0,cpu_array0 , size_in_bytes, hipMemcpyHostToDevice );
    hipMemcpy( gpu_array1,cpu_array1 , size_in_bytes, hipMemcpyHostToDevice );
	
	/* Execute 4 simple math operation*/ 
	for(int kernel=0; kernel<4; kernel++)
    {
      switch(kernel)
      {
            case 0:{ 
					  execute_gpu_sharedmem_arrayAdd(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                    } break;                                                                                     
            case 1:{
					  execute_gpu_arraySubtract(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                   }break;                                     
           case 2:{
					  execute_gpu_arrayMult(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                   }break;                                                                 
           case 3:{      
					  execute_gpu_arrayMod(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                   }break;                                                                   
            default: exit(1); break;                                                                                                         
      }	
	}	
		
	/*Free the arrays on the CPU*/
	free(cpu_array0);
	free(cpu_array1);
	free(cpu_array_res);
    /* Free the arrays on the GPU as now we're done with them */
    hipFree(gpu_array0);
	hipFree(gpu_array1);
    hipFree(gpu_arrayresult);	
	//Destroy all allocations and reset all state on the current device in the current process
	hipDeviceReset();
}

int main(int argc, char** argv)
{
	// read command line arguments
	int totalThreads = (1 << 20);
	int blockSize = 256;
	
	if (argc >= 2) {
		totalThreads = atoi(argv[1]);
	}
	if (argc >= 3) {
		blockSize = atoi(argv[2]);
	}

	int numBlocks = totalThreads/blockSize;
	
	/* code check to make sure blockSize is multiple of WARP */
    if(blockSize<WARP)
	{                        
           blockSize=WARP;
           printf("Warning: Block size specified is less than size of WARP.It got modified to be: %i\n",WARP);     
    }
    else
	{
            if(blockSize % WARP!=0)
            {
                    blockSize=(blockSize+0.5*WARP)/WARP*WARP;
                    printf("Warning: Block size specified is not evenly divisible by the size of WARP.\n");
                    printf("It got modified to be the nearst number that can be evenly divisible by the size of WARP.\n");
                    printf("Now, the blocksize is:%i\n",blockSize);     
            }
    }

	/* code check to make sure Total number of threads is multiple of blockSize*/
	if (totalThreads % blockSize != 0) {
		++numBlocks;
		totalThreads = numBlocks*blockSize;
		
		printf("Warning: Total thread count is not evenly divisible by the block size\n");
		printf("The total number of threads will be rounded up to %d\n", totalThreads);
	}
	//execute_gpu_global_test(numBlocks,blockSize); //  test harness for executing kernel using global memory
	execute_gpu_shared_test(numBlocks,blockSize); //  test harness for executing kernel using shared memory
	//execute_gpu_const_test(numBlocks,blockSize);  //  test harness for executing kernel using constant memory
     
	
}
