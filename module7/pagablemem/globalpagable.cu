#include "hip/hip_runtime.h"
#include <stdio.h>
#include "globalpagable.h"
#include "utility.h"
#include "globalmacro.h"

/* Actual kernels that use the global memory*/					
static __global__ void arrayAdd(int *array0,int *array1,int* arrayres);
static __global__ void arraySubtract(int *array0,int *array1,int* arrayres);
static __global__ void arrayMult(int *array0,int *array1,int* arrayres);
static __global__ void arrayMod(int *array0,int *array1,int* arrayres);

__global__ void arrayAdd(int *array0,int *array1,int* arrayres) {
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;// collapse the higher dimension layout or nested layout down to flat 2D
	const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;// collapse flat 2D down to 1D, whose index is global thread index   
    if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
	    for(int i=0;i<MAXOPERIONS;i++)
		{
			arrayres[global_idx]=array0[global_idx]+array1[global_idx];
		}
    }
}
__global__ void arraySubtract(int *array0,int *array1,int* arrayres) {
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x; // collapse the higher dimension layout or nested layout down to flat 2D
	const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx; // collapse flat 2D down to 1D, whose index is global thread index

	if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
		for(int i=0;i<MAXOPERIONS;i++)
		{
			arrayres[global_idx]=array0[global_idx]-array1[global_idx];
		}
	}
}
__global__ void arrayMult(int *array0,int *array1,int* arrayres) {

    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;    // collapse the higher dimension layout or nested layout down to flat 2D
    const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;


    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;    // collapse flat 2D down to 1D, whose index is global thread index
    if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
	    for(int i=0;i<MAXOPERIONS;i++)
		{
			arrayres[global_idx]=array0[global_idx]*array1[global_idx];
		}
    }
}
__global__ void arrayMod(int *array0,int *array1,int* arrayres) {
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;    // collapse the higher dimension layout or nested layout down to flat 2D
    const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;    // collapse flat 2D down to 1D, whose index is global thread index
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;
    if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
	    for(int i=0;i<MAXOPERIONS;i++)
		{
			arrayres[global_idx]=array0[global_idx]%array1[global_idx];
		}
    }
}

__host__ void execute_gpu_arrayAdd(int numBlocks,int blockSize,int *const gpu_array0,int * const gpu_array1,int* const gpu_arrayresult,int* const cpu_array_res){ 	
    int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	const dim3 threads_layout(WARP,blockSize/WARP); // there are multiple ways of layout to achieve blocksize. I choose to fix the  blockDim.x as the WARP size
    const dim3 blocks_layout(1,numBlocks);// there are multiple ways of layout to achieve numBlocks, I choose to fix the gridDim.x to 1
	hipEvent_t kernel_start1, kernel_stop1;
	float delta_time1 = 0.0f;
	hipEventCreate(&kernel_start1);
	hipEventCreateWithFlags(&kernel_stop1,hipEventBlockingSync);
	hipEventRecord(kernel_start1, 0);//0 is the default stream
	arrayAdd<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult); // kernel call to add two 2-D arrays 
	hipEventRecord(kernel_stop1, 0);
	hipEventSynchronize(kernel_stop1);
	hipEventElapsedTime(&delta_time1, kernel_start1,kernel_stop1);	
	hipMemcpy(cpu_array_res, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
	printf("Kernel 0 (Add) is called! \n");printf("The Kernel 0 (Add) performs the math operation %d times! \n",MAXOPERIONS);
	if(VERBOSE){printf("Array Result:\n");print_array(cpu_array_res,cpu_arr_size_y,cpu_arr_size_x);}//debug only
	printf("GPU execution with global mem takes: %.3fms\n",delta_time1);printf("*******\n");
	hipEventDestroy(kernel_start1);hipEventDestroy(kernel_stop1);
}
__host__ void execute_gpu_arraySubtract(int numBlocks,int blockSize,int *const gpu_array0,int * const gpu_array1,int* const gpu_arrayresult,int* const cpu_array_res){
	int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	const dim3 threads_layout(WARP,blockSize/WARP); // there are multiple ways of layout to achieve blocksize. I choose to fix the  blockDim.x as the WARP size
    const dim3 blocks_layout(1,numBlocks);// there are multiple ways of layout to achieve numBlocks, I choose to fix the gridDim.x to 1
	hipEvent_t kernel_start1, kernel_stop1;
	float delta_time1 = 0.0f;
	hipEventCreate(&kernel_start1);
	hipEventCreateWithFlags(&kernel_stop1,hipEventBlockingSync);
	hipEventRecord(kernel_start1, 0);//0 is the default stream
	arraySubtract<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult);//kernel call to subtract two 2-D arrays 
	hipEventRecord(kernel_stop1, 0);
	hipEventSynchronize(kernel_stop1);
	hipEventElapsedTime(&delta_time1, kernel_start1,kernel_stop1);
	hipMemcpy(cpu_array_res, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
	printf("Kernel 1 (subtract) is called! \n");printf("The Kernel 1 (subtract) performs the math operation %d times! \n",MAXOPERIONS);
	if(VERBOSE){printf("Array Result:\n");print_array(cpu_array_res,cpu_arr_size_y,cpu_arr_size_x);}
	printf("GPU execution with global mem takes: %.3fms\n",delta_time1);printf("*******\n");
	hipEventDestroy(kernel_start1);hipEventDestroy(kernel_stop1);
}
__host__ void execute_gpu_arrayMult(int numBlocks,int blockSize,int *const gpu_array0,int * const gpu_array1,int* const gpu_arrayresult,int* const cpu_array_res){
    int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	const dim3 threads_layout(WARP,blockSize/WARP); // there are multiple ways of layout to achieve blocksize. I choose to fix the  blockDim.x as the WARP size
    const dim3 blocks_layout(1,numBlocks);// there are multiple ways of layout to achieve numBlocks, I choose to fix the gridDim.x to 1
	hipEvent_t kernel_start1, kernel_stop1;
	float delta_time1 = 0.0f;
	hipEventCreate(&kernel_start1);
	hipEventCreateWithFlags(&kernel_stop1,hipEventBlockingSync);
	hipEventRecord(kernel_start1, 0);//0 is the default stream
	arrayMult<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult);//kernel call to (elementwise)multiply two 2-D arrays 
	hipEventRecord(kernel_stop1, 0);//0 is the default stream
	hipEventSynchronize(kernel_stop1);
	hipEventElapsedTime(&delta_time1, kernel_start1,kernel_stop1);
	hipMemcpy(cpu_array_res, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
	printf("Kernel 2 (multiplication) is called! \n");printf("Kernel 2 (multiplication) performs the math operation %d times! \n",MAXOPERIONS);
	if(VERBOSE){printf("Array Result:\n");print_array(cpu_array_res,cpu_arr_size_y,cpu_arr_size_x); }
	printf("GPU execution with global mem takes: %.3fms\n",delta_time1);printf("*******\n");
	hipEventDestroy(kernel_start1);hipEventDestroy(kernel_stop1);	
}
__host__ void execute_gpu_arrayMod(int numBlocks,int blockSize,int *const gpu_array0,int * const gpu_array1,int* const gpu_arrayresult,int* const cpu_array_res){
    int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	const dim3 threads_layout(WARP,blockSize/WARP); // there are multiple ways of layout to achieve blocksize. I choose to fix the  blockDim.x as the WARP size
    const dim3 blocks_layout(1,numBlocks);// there are multiple ways of layout to achieve numBlocks, I choose to fix the gridDim.x to 1
	hipEvent_t kernel_start1, kernel_stop1;
	float delta_time1 = 0.0f;
	hipEventCreate(&kernel_start1);
	hipEventCreateWithFlags(&kernel_stop1,hipEventBlockingSync);
	hipEventRecord(kernel_start1, 0);//0 is the default stream
	arrayMod<<<blocks_layout,threads_layout>>>(gpu_array0,gpu_array1,gpu_arrayresult);//kernel call to (elementwise) mod divide two 2-D arrays
	hipEventRecord(kernel_stop1, 0);//0 is the default stream
	hipEventSynchronize(kernel_stop1);
	hipEventElapsedTime(&delta_time1, kernel_start1,kernel_stop1);						
	hipMemcpy(cpu_array_res, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu 
	printf("Kernel 3 (mod) is called! \n");printf("Kernel 3 (mod) performs the math operation %d times! \n",MAXOPERIONS);
	if(VERBOSE){printf("Array Result:\n");print_array(cpu_array_res,cpu_arr_size_y,cpu_arr_size_x);}// debug only
	printf("GPU execution with global mem takes: %.3fms\n",delta_time1);printf("*******\n");
	hipEventDestroy(kernel_start1);hipEventDestroy(kernel_stop1);	
}

void execute_gpu_global_test(int numBlocks, int blockSize){
	printf("Unit Test 1: Simple Math Operations with global memory\n");
	printf("-------------------------------------------------------\n");
	int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	int *cpu_array0,*cpu_array1,*cpu_array_res; 
	cpu_array0 = (int *) malloc(size_in_bytes);//pagable
	cpu_array1 = (int *)malloc(size_in_bytes);
	cpu_array_res = (int *)malloc(size_in_bytes);
    cpu_array0_int(cpu_array0,cpu_arr_size_y,cpu_arr_size_x);
	cpu_array1_int(cpu_array1,cpu_arr_size_y,cpu_arr_size_x);
	if(VERBOSE){//print out the arrays for debuging
		printf("The following two arrays are initialized on cpu! \n");
		printf("Array0:\n");print_array(cpu_array0,cpu_arr_size_y,cpu_arr_size_x);
		printf("Array1:\n");print_array(cpu_array1,cpu_arr_size_y,cpu_arr_size_x);
	}
    int * gpu_array0, * gpu_array1,*gpu_arrayresult;
	hipMalloc((void **)&gpu_array0, size_in_bytes);
	hipMalloc((void **)&gpu_array1, size_in_bytes);
    hipMalloc((void **)&gpu_arrayresult, size_in_bytes);
	hipMemcpy( gpu_array0,cpu_array0 , size_in_bytes, hipMemcpyHostToDevice );
    hipMemcpy( gpu_array1,cpu_array1 , size_in_bytes, hipMemcpyHostToDevice ); 
	for(int kernel=0; kernel<4; kernel++){//Execute 4 simple math operation
      switch(kernel){
            case 0:{ execute_gpu_arrayAdd(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                    } break;                                                                                     
            case 1:{execute_gpu_arraySubtract(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                   }break;                                     
           case 2:{execute_gpu_arrayMult(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                   }break;                                                                 
           case 3:{ execute_gpu_arrayMod(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                   }break;                                                                   
            default: exit(1); break;}	
	}	
	free(cpu_array0);free(cpu_array1);free(cpu_array_res);
    hipFree(gpu_array0);hipFree(gpu_array1);hipFree(gpu_arrayresult);	
	hipDeviceReset();//Destroy all allocations and reset all state on the current device in the current process
}