#include "hip/hip_runtime.h"
#include <stdio.h>
#include "sharedmem.h"
#include "utility.h"
#include "globalmacro.h"

/*kernels that use the shared memory*/
__global__ void gpu_arrayAdd_shared(int *array0,int *array1,int* arrayres,int num_elements,int totalnumofThreads);
__global__ void gpu_arraySubtract_shared(int *array0,int *array1,int* arrayres,int num_elements,int totalnumofThreads);
__global__ void gpu_arrayMult_shared(int *array0,int *array1,int* arrayres,int num_elements,int totalnumofThreads);
__global__ void gpu_arrayMod_shared(int *array0,int *array1,int* arrayres,int num_elements,int totalnumofThreads);

__global__ void gpu_arrayAdd_shared(int *array0,int *array1,int* arrayres) {
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;// collapse the higher dimension layout or nested layout down to flat 2D
	const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx; // collapse flat 2D down to 1D, whose index is global thread index
	 extern __shared__ int shared_tmp[];// total dynamically  allocated shared mem // 49KB limit
	 int *arry0shared=shared_tmp;
	 int *arry1shared=(int*)&shared_tmp[blockDim.x];
	 copy_data_to_shared(array0,arry0shared,global_idx,threadIdx.x);//array0-->arry0shared
	 copy_data_to_shared(array1,arry1shared,global_idx,threadIdx.x);//array1-->arry1shared
	if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
	    for(int i=0;i<MAXOPERIONS;i++){
			arrayres[global_idx]=arry0shared[threadIdx.x]+arry1shared[threadIdx.x];
		}
    }
}
__global__ void gpu_arraySubtract_shared(int *array0,int *array1,int* arrayres) {
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;    // collapse the higher dimension layout or nested layout down to flat 2D
	const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
	

    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;    // collapse flat 2D down to 1D, whose index is global thread index
	 extern __shared__ int shared_tmp[];// total dynamically  allocated shared mem // 49KB limit
	 int *arry0shared=shared_tmp;
	 int *arry1shared=(int*)&shared_tmp[blockDim.x];
	 copy_data_to_shared(array0,arry0shared,global_idx,threadIdx.x);//array0-->arry0shared
	 copy_data_to_shared(array1,arry1shared,global_idx,threadIdx.x);//array1-->arry1shared
	if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
	    for(int i=0;i<MAXOPERIONS;i++)
		{
			arrayres[global_idx]=arry0shared[threadIdx.x]-arry1shared[threadIdx.x];
		}
    }

}    
__global__ void gpu_arrayMult_shared(int *array0,int *array1,int* arrayres) {
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;    // collapse the higher dimension layout or nested layout down to flat 2D
	const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;    // collapse flat 2D down to 1D, whose index is global thread index
	 extern __shared__ int shared_tmp[];// total dynamically  allocated shared mem // 49KB limit
	 int *arry0shared=shared_tmp;
	 int *arry1shared=(int*)&shared_tmp[blockDim.x];
	 copy_data_to_shared(array0,arry0shared,global_idx,threadIdx.x);//array0-->arry0shared
	 copy_data_to_shared(array1,arry1shared,global_idx,threadIdx.x);//array1-->arry1shared
	if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
	    for(int i=0;i<MAXOPERIONS;i++)
		{
			arrayres[global_idx]=arry0shared[threadIdx.x]*arry1shared[threadIdx.x];
		}
    }
}
__global__ void gpu_arrayMod_shared(int *array0,int *array1,int* arrayres) {
    const int idx = (blockIdx.x * blockDim.x) + threadIdx.x;  // collapse the higher dimension layout or nested layout down to flat 2D
	const int idy = (blockIdx.y * blockDim.y) + threadIdx.y;
    const int global_idx = ((gridDim.x * blockDim.x) * idy) + idx;   // collapse flat 2D down to 1D, whose index is global thread index
	 extern __shared__ int shared_tmp[];// total dynamically  allocated shared mem // 49KB limit
	 int *arry0shared=shared_tmp;
	 int *arry1shared=(int*)&shared_tmp[blockDim.x];
	 copy_data_to_shared(array0,arry0shared,global_idx,threadIdx.x);//array0-->arry0shared
	 copy_data_to_shared(array1,arry1shared,global_idx,threadIdx.x);//array1-->arry1shared	 
	if(idx<(gridDim.x*blockDim.x) && idy<(gridDim.y*blockDim.y)){
	    for(int i=0;i<MAXOPERIONS;i++)
		{
			arrayres[global_idx]=arry0shared[threadIdx.x]%arry1shared[threadIdx.x];
		}
    }
}

__host__ void execute_gpu_sharedmem_arrayAdd(int numBlocks,int blockSize,int *const gpu_array0,int * const gpu_array1,int* const gpu_arrayresult,int* const cpu_array_res){
	int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	const dim3 threads_layout(blockSize,1); // using 1 row, blockSize colmns layout 
    const dim3 blocks_layout(numBlocks,1);// there are multiple ways of layout to achieve numBlocks, I choose to fix the  1 row, numBlocks colmn layout
	hipEvent_t kernel_start1, kernel_stop1;
	float delta_time1 = 0.0f;
	hipEventCreate(&kernel_start1);
	hipEventCreateWithFlags(&kernel_stop1,hipEventBlockingSync);
	hipEventRecord(kernel_start1, 0);//0 is the default stream
	gpu_arrayAdd_shared<<<blocks_layout,threads_layout,blockSize*3*sizeof(int)>>>(gpu_array0,gpu_array1,gpu_arrayresult);
	hipEventRecord(kernel_stop1, 0);
	hipEventSynchronize(kernel_stop1);
	hipEventElapsedTime(&delta_time1, kernel_start1,kernel_stop1);	
	hipMemcpy(cpu_array_res, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
	printf("Kernel 0 (Add) is called! \n");printf("The Kernel 0 (Add) performs the math operation %d times! \n",MAXOPERIONS);
	if(VERBOSE){printf("Array Result:\n");print_array(cpu_array_res,cpu_arr_size_y,cpu_arr_size_x);}//debug only
	printf("GPU execution with shared mem takes: %.3fms\n",delta_time1);printf("*******\n");;
	hipEventDestroy(kernel_start1);hipEventDestroy(kernel_stop1);
}
__host__ void execute_gpu_sharedmem_arraySubtract(int numBlocks,int blockSize,int *const gpu_array0,int * const gpu_array1,int* const gpu_arrayresult,int* const cpu_array_res){
	int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	const dim3 threads_layout(blockSize,1); // 
    const dim3 blocks_layout(numBlocks,1);// 
	hipEvent_t kernel_start1, kernel_stop1;
	float delta_time1 = 0.0f;
	hipEventCreate(&kernel_start1);
	hipEventCreateWithFlags(&kernel_stop1,hipEventBlockingSync);
	hipEventRecord(kernel_start1, 0);//0 is the default stream
	gpu_arraySubtract_shared<<<blocks_layout,threads_layout,blockSize*2*sizeof(int)>>>(gpu_array0,gpu_array1,gpu_arrayresult);
	hipEventRecord(kernel_stop1, 0);
	hipEventSynchronize(kernel_stop1);
	hipEventElapsedTime(&delta_time1, kernel_start1,kernel_stop1);
	hipMemcpy(cpu_array_res, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
	printf("Kernel 1 (subtract) is called! \n");printf("The Kernel 1 (subtract) performs the math operation %d times! \n",MAXOPERIONS);
	if(VERBOSE){printf("Array Result:\n");print_array(cpu_array_res,cpu_arr_size_y,cpu_arr_size_x);					 }
	printf("GPU execution with shared mem takes: %.3fms\n",delta_time1);printf("*******\n");
	hipEventDestroy(kernel_start1);hipEventDestroy(kernel_stop1);
}
__host__ void execute_gpu_sharedmem_arrayMult(int numBlocks,int blockSize,int *const gpu_array0,int * const gpu_array1,int* const gpu_arrayresult,int* const cpu_array_res){
	int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	const dim3 threads_layout(blockSize,1); // 
    const dim3 blocks_layout(numBlocks,1);// 
	hipEvent_t kernel_start1, kernel_stop1;
	float delta_time1 = 0.0f;
	hipEventCreate(&kernel_start1);
	hipEventCreateWithFlags(&kernel_stop1,hipEventBlockingSync);
	hipEventRecord(kernel_start1, 0);//0 is the default stream
	gpu_arrayMult_shared<<<blocks_layout,threads_layout,blockSize*2*sizeof(int)>>>(gpu_array0,gpu_array1,gpu_arrayresult);
	hipEventRecord(kernel_stop1, 0);//0 is the default stream
	hipEventSynchronize(kernel_stop1);
	hipEventElapsedTime(&delta_time1, kernel_start1,kernel_stop1);
	hipMemcpy(cpu_array_res, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); // memcopy from gpu to cpu
	printf("Kernel 2 (multiplication) is called! \n");printf("Kernel 2 (multiplication) performs the math operation %d times! \n",MAXOPERIONS);
	if(VERBOSE){printf("Array Result:\n");print_array(cpu_array_res,cpu_arr_size_y,cpu_arr_size_x); }//debug only
	printf("GPU execution with shared mem takes: %.3fms\n",delta_time1);printf("*******\n");
	hipEventDestroy(kernel_start1);hipEventDestroy(kernel_stop1);	
}
__host__ void execute_gpu_sharedmem_arrayMod(int numBlocks,int blockSize,int *const gpu_array0,int * const gpu_array1,int* const gpu_arrayresult,int* const cpu_array_res){
	int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	const dim3 threads_layout(blockSize,1); 
    const dim3 blocks_layout(numBlocks,1);
	hipEvent_t kernel_start1, kernel_stop1;
	float delta_time1 = 0.0f;
	hipEventCreate(&kernel_start1);
	hipEventCreateWithFlags(&kernel_stop1,hipEventBlockingSync);
	hipEventRecord(kernel_start1, 0);//0 is the default stream
	gpu_arrayMod_shared<<<blocks_layout,threads_layout,blockSize*2*sizeof(int)>>>(gpu_array0,gpu_array1,gpu_arrayresult);
	hipEventRecord(kernel_stop1, 0);//0 is the default stream
	hipEventSynchronize(kernel_stop1);
	hipEventElapsedTime(&delta_time1, kernel_start1,kernel_stop1);						
	hipMemcpy(cpu_array_res, gpu_arrayresult, size_in_bytes, hipMemcpyDeviceToHost); 
	printf("Kernel 3 (mod) is called! \n");printf("Kernel 3 (mod) performs the math operation %d times! \n",MAXOPERIONS);
	if(VERBOSE){printf("Array Result:\n");print_array(cpu_array_res,cpu_arr_size_y,cpu_arr_size_x);}
	printf("GPU execution with shared mem takes: %.3fms\n",delta_time1);printf("*******\n");
	hipEventDestroy(kernel_start1);hipEventDestroy(kernel_stop1);	
}

void execute_gpu_shared_test(int numBlocks, int blockSize){
    printf("Unit Test2: Simple Math Operations with shared memory\n");
    printf("-------------------------------------------------------\n");
	int totalThreads=numBlocks*blockSize;
	int cpu_arr_size_y=1;//row
	int cpu_arr_size_x=totalThreads;//column
	int size_in_bytes = cpu_arr_size_x* cpu_arr_size_y* sizeof(int);
	/* dynamically allocate the memory on the host*/
	int *cpu_array0,*cpu_array1,*cpu_array_res; 
	cpu_array0 = (int *) malloc(size_in_bytes);//pagable
	cpu_array1 = (int *)malloc(size_in_bytes);
	cpu_array_res = (int *)malloc(size_in_bytes);
    cpu_array0_int(cpu_array0,cpu_arr_size_y,cpu_arr_size_x);
	cpu_array1_int(cpu_array1,cpu_arr_size_y,cpu_arr_size_x);
	if(VERBOSE){//print out the arrays for debuging
		printf("The following two arrays are initialized on cpu! \n");
		printf("Array0:\n");print_array(cpu_array0,cpu_arr_size_y,cpu_arr_size_x);
		printf("Array1:\n");print_array(cpu_array1,cpu_arr_size_y,cpu_arr_size_x);
	}
	 /* Device memory allocation */
    int * gpu_array0, * gpu_array1,*gpu_arrayresult;
	hipMalloc((void **)&gpu_array0, size_in_bytes);
	hipMalloc((void **)&gpu_array1, size_in_bytes);
    hipMalloc((void **)&gpu_arrayresult, size_in_bytes);
	hipMemcpy( gpu_array0,cpu_array0 , size_in_bytes, hipMemcpyHostToDevice );
    hipMemcpy( gpu_array1,cpu_array1 , size_in_bytes, hipMemcpyHostToDevice );
	for(int kernel=0; kernel<4; kernel++){//Execute 4 simple math operation 
      switch(kernel){
            case 0:{ execute_gpu_sharedmem_arrayAdd(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                    } break;                                                                                     
            case 1:{execute_gpu_sharedmem_arraySubtract(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                   }break;                                     
           case 2:{execute_gpu_sharedmem_arrayMult(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                   }break;                                                                 
           case 3:{execute_gpu_sharedmem_arrayMod(numBlocks,blockSize,gpu_array0,gpu_array1,gpu_arrayresult,cpu_array_res);
                   }break;                                                                   
            default: exit(1); break;                                                                                                         
      }	
	}	
	free(cpu_array0);
	free(cpu_array1);
	free(cpu_array_res);
    hipFree(gpu_array0);
	hipFree(gpu_array1);
    hipFree(gpu_arrayresult);	
	hipDeviceReset();	//Destroy all allocations and reset all state on the current device in the current process
}
